#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

#define M 20

__global__ void add(int *A, int *B, int *C) {
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}
int main() {
	int i, *A, *B, *C;
	A = (int *) malloc(M * sizeof(int));
	B = (int *) malloc(M * sizeof(int));
	C = (int *) malloc(M * sizeof(int));
	for (i = 0; i < M; i++) {
		*(A + i) = i;
		*(B + i) = 2 * i;
	}
	int *dev_A, *dev_B, *dev_C;
	int size = sizeof(int);
	// allocate device copies of a, b, c
	hipMalloc((void**) &dev_A, size * M);
	hipMalloc((void**) &dev_B, size * M);
	hipMalloc((void**) &dev_C, size * M);
	hipMemcpy(dev_A, A, size * M, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, size * M, hipMemcpyHostToDevice);
	// launch add() kernel on GPU, passing parameters
	add<<<1, M>>>(dev_A, dev_B, dev_C);
	// copy device result back to host copy of c
	hipMemcpy(C, dev_C, size * M, hipMemcpyDeviceToHost);
	for (i = 0; i < M; i++)
		printf("%d\n", C[i]);
	free(A);
	free(B);
	free(C);
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	return 0;
}
